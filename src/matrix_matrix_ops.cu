#include "hip/hip_runtime.h"
#include "matrix_matrix_ops_cuda.hpp"

void fun_q_sparse_product(int n, 
int m, 
int q_nnz, 
int* q_i, 
int* q_j, 
double* q_v, 
int a_nnz, 
int* a_i, 
int* a_j, 
double* a_v, 
double* h_v, 
double* out) 
{
    int num_blocks;
    int block_size = 512;
    num_blocks = (n + block_size - 1) / block_size;
    q_sparse_product<<<num_blocks, block_size>>>(n, 
        m, 
        q_nnz, 
        q_i, 
        q_j, 
        q_v, 
        a_nnz, 
        a_i, 
        a_j, 
        a_v, 
        h_v, 
        out);
}

__global__ void q_sparse_product(int n, 
int m, 
int q_nnz, 
int* q_i, 
int* q_j, 
double* q_v, 
int a_nnz, 
int* a_i, 
int* a_j, 
double* a_v, 
double* h_v, 
double* out) 
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  double total = 0.0;
  int row_offset = 0;
}